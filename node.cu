#include "hip/hip_runtime.h"
#include <unistd.h> 
#include <sys/socket.h> 
#include <stdlib.h> 
#include <netinet/in.h> 
#include <arpa/inet.h>
#include <string.h> 
#include <iostream>
#include <cstdlib>
#include <math.h>
#include "stdio.h"
#include <assert.h>
#include <iomanip>
#include <chrono>
#include <sys/ipc.h> 
#include <sys/shm.h> 

#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include <thread>
#include "ipc.h"
#include <matrix.h>

#ifndef BLAS
///////////////////////// KERNEL FUNCTION ////////////////////////////////
__global__ void
matmul
(float *a, float *b, float *c,
size_t A_row, size_t A_col, size_t B_col){
  int row = blockIdx.x * blockDim.x +  threadIdx.x;
  int col = blockIdx.y * blockDim.y +  threadIdx.y;
  if (row >= A_row || col >= B_col) return;
  float sum =0;

//  printf("%d  %d  %d\n", row , col, (A_col-1)*B_col + col);
  for (int i = 0; i < A_col; ++i){
    sum += a[(row) * A_col + i] * b[i*(B_col) + col];
  }
  c[row*(A_row) + col] = sum;
}

////////////////////////// KERNEL FUNCTION ////////////////////////////////
#endif


using namespace std;

void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{   
    for (unsigned int i = 0; i < hA; ++i)
      for (unsigned int j = 0; j < wB; ++j)
        {   
            double sum = 0;
            
            for (unsigned int k = 0; k < wA; ++k)
            {   
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            
            C[i * wB + j] = (float)sum;
        }
}


#define X 1000
#define Y 1000
#define Z 1000
#undef VERIFY

Matrix A(X, Y, false, 1);
Matrix B(Y, Z, false, 2);
Matrix C(X,Z);

int launch_kernel(){

  A.toCUDA();
  B.toCUDA();
  C.cuda_malloc();
#ifdef BLAS
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  hipblasHandle_t handle;
  checkCudaErrors(hipblasCreate(&handle));
  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, X, Y, Z, &alpha, A.data_cuda, X, B.data_cuda, Y, &beta, C.data_cuda, Z));
        checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));
  
#else
  dim3 dimGrid(1, 1);
  dim3 dimBlock(X, Z);
  if (X*Z > 2014){
    dimBlock.x = 32;
    dimBlock.y = 32;
    dimGrid.x = ceil(double(X)/double(dimBlock.x));
    dimGrid.y = ceil(double(Z)/double(dimBlock.y));
  }
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  matmul<<<dimGrid,dimBlock, 0, stream1>>>(A.data_cuda, B.data_cuda, C.data_cuda, X, Y, Z);
#endif

  C.fromCUDA(NULL);
  key_t key = ftok("res",1234);
  int shmid = shmget(key, X*Z*sizeof(float), 0660 | IPC_CREAT); 
  std::cout <<shmid <<" and "<< errno<<std::endl;
  float *data = (float*) shmat(shmid,(void*)0,0); 

  memcpy(data, C.cuda_res, X*Z*sizeof(float));
  shmdt(data);

  //C.printGPU();
  std::cout << "GPU IS DONE ! " << std::endl;
  int device;
   hipGetDevice(&device);
 #ifdef VERIFY
   matrixMulCPU(C.data_cpu, A.data_cpu, B.data_cpu, X, Y, Z);
   std::cout << "CPU IS DONE ! " << std::endl;
  // std::cout <<   props.maxThreadsDim[0];
  // C.printCPU();
   std::cout << "ret val is: " <<hipGetLastError() << " and the answer is: " << C.verify() << std::endl;;
#endif
   A.freeDeviceMem();
   B.freeDeviceMem();
   C.freeDeviceMem();
   C.freeHostMem();
   return 0;
}


int connect_to_master(){
    int sockfd = 0; 
    struct sockaddr_in serv_addr; 

    if ((sockfd = socket(AF_INET, SOCK_STREAM, 0)) < 0) 
    { 
        printf("\n Socket creation error \n"); 
        return -1; 
    } 
   
    memset(&serv_addr, 0, sizeof(serv_addr)); 
   
    serv_addr.sin_family = AF_INET; 
    serv_addr.sin_port = htons(PORT); 
       
    // Convert IPv4 and IPv6 addresses from text to binary form 
    if(inet_pton(AF_INET, "127.0.0.1", &serv_addr.sin_addr)<=0)  
    { 
        printf("\nInvalid address/ Address not supported \n"); 
        return -1; 
    } 
   
    if (connect(sockfd, (struct sockaddr *)&serv_addr, sizeof(serv_addr)) < 0) 
    { 
        printf("\nConnecntion Failed \n"); 
        return -1; 
    } 

    IPCCommand request = NODE_CONNECT;
    send(sockfd, &request, sizeof(IPCCommand) , 0 ); 
    std::cout << "First Request message sent\n";

    IPCCommand response;
    read( sockfd , &response, sizeof(IPCCommand));
    if(response != MASTER_ACK){
      return -1;
    }

    std::cout << "Connection accepted for GPU 0\n"; 
    return sockfd; 
}

void main_event_loop(int socketfd) { 
    IPCCommand from_master, to_master;
    int val_read = read(socketfd, &from_master, sizeof(IPCCommand));
    while (val_read >0 && from_master != MASTER_NODE_SHUTDOWN){
      launch_kernel();
      to_master = NODE_OUTPUT_AVAILABLE;
      send(socketfd, &to_master, sizeof(IPCCommand),0);
      val_read = read(socketfd, &from_master, sizeof(IPCCommand));
    }
}


int send_fin(int sockfd){
    std::string finish = "Finished";
    send(sockfd, finish.c_str() , strlen(finish.c_str()) , 0 ); 
    printf("Finished!\n"); 
    char buffer[1024] = {0};
    int valread = read( sockfd , buffer, 1024); 
    printf("%s\n",buffer );
    return 0;
}


int main(){
    int node_socket = connect_to_master();
    main_event_loop(node_socket);
    send_fin(node_socket);
}
